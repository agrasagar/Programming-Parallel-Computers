#include "hip/hip_runtime.h"
#include "cp.h"
#include <iostream>     // std::cout
#include <algorithm>    // std::for_each
#include <vector>       // std::vector
#include <cmath>
#define TILE 32

__global__ void matrixMul(double *Md,double *Mtd, float *Rd, int ny, int nx){
	
	int x = blockIdx.x*TILE + threadIdx.x;
	int y = blockIdx.y*TILE + threadIdx.y;
	__shared__ float tile_A[TILE][TILE];
	__shared__ float tile_B[TILE][TILE];
	int i;
	float sum = 0.0;
	for(i = 0; i< ((nx-1)/TILE)+1; ++i ){
		
		//Put tile_A on shared memory
		if((i*TILE + threadIdx.x )< nx && y < ny ){
			tile_A[threadIdx.y][threadIdx.x] = Md[y*nx + i*TILE + threadIdx.x];
		}
		else{
			tile_A[threadIdx.y][threadIdx.x] = 0;
		}
		//Put tile_B on shared memory
		if((i*TILE + threadIdx.y)< nx && x < ny ){
			tile_B[threadIdx.y][threadIdx.x] = Mtd[(i*TILE + threadIdx.y)*ny + x];
		}
		else{
			tile_B[threadIdx.y][threadIdx.x] = 0;
		}
		__syncthreads();
		
		for(int j = 0; j < TILE; ++j){
			sum += tile_A[threadIdx.y][j] * tile_B[j][threadIdx.x];
		}
		__syncthreads();
	}
	
	if(x >= ny || y >= ny || x<y){
		return;
	}
	
	Rd[((blockIdx.y*blockDim.y + threadIdx.y)*ny) + (blockIdx.x*blockDim.x) + threadIdx.x] = sum;
}

void correlate(int ny, int nx, const float* data, float* result) {
		
	double* dataVec;
	double* dataVecT;
	hipHostMalloc( (void **) &dataVec, sizeof(double)*nx*ny);
	hipHostMalloc( (void **) &dataVecT, sizeof(double)*ny*nx);
	int y, rowStart, rowEnd;
	int i;
	double meanOfEachRow, meanOfSquareRoot,num;
	std::vector<double> v(nx), vzeroSquaremean(nx);
	for(y = 0; y< ny ; ++y){        //traversing through y axis
		rowStart = y*nx;
		rowEnd = rowStart + nx;
		meanOfEachRow = std::accumulate(data+rowStart, data+rowEnd, 0.0)/nx;
		std::transform(data+rowStart, data+rowEnd, v.begin(), [&meanOfEachRow](double val){ return (val - meanOfEachRow); });
		std::transform(v.begin(), v.end(), vzeroSquaremean.begin(), [](double val){ return std::pow(val, 2); });
		meanOfSquareRoot = std::sqrt(std::accumulate(vzeroSquaremean.begin(), vzeroSquaremean.end(), 0.0));
		for(i = 0; i < nx; ++i) {
			num = v[i]/meanOfSquareRoot;
			dataVec[i+y*nx] = num;
		}
	}
	//Matrix transposel
	for( y = 0 ; y <ny ; ++y){
		for(i = 0 ; i < nx; ++i){
			dataVecT[i*ny + y] = dataVec[y*nx + i];
		}
	} 
	//Matrix multiplication
	//matrix nx*ny
	//matrixT ny*nx
	//after multiplication product matrix would be of size ny*ny
	dim3 dimBlock(TILE,TILE);
	dim3 dimGrid(std::ceil((double)ny/dimBlock.x), std::ceil((double)ny/(double)dimBlock.y));
	int sizeM = nx*ny*sizeof(double);
	int sizeR = ny*ny*sizeof(float);
	double* Md = 0;
	double* Mtd = 0; 
	float* Rd = 0;
	//Allocate Md and Pd on device
	hipMalloc((void **)&Md, sizeM);
	hipMemcpy(Md, dataVec, sizeM, hipMemcpyHostToDevice);
	hipMalloc((void **)&Mtd, sizeM);
        hipMemcpy(Mtd, dataVecT, sizeM, hipMemcpyHostToDevice);

	hipMalloc((void **)&Rd, sizeR);
	matrixMul<<<dimGrid, dimBlock>>>(Md,Mtd,Rd,ny, nx);
	hipMemcpy(result, Rd, sizeR, hipMemcpyDeviceToHost);

	hipFree(Md);
	hipFree(Mtd);
	hipFree(Rd);
	hipFree(dataVec);
	hipFree(dataVecT);	
}
