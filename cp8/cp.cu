#include "hip/hip_runtime.h"
#include "cp.h"
#include <iostream>     // std::cout
#include <algorithm>    // std::for_each
#include <vector>       // std::vector
#include <cmath>
__global__ void matrixMul(double *Md, float *Rd, int ny, int nx){
	
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	double sum = 0;	
	if(x >= ny || y >= ny || x<y){
		return;
	}
	for(int k= 0; k<nx ; k++){
		sum += Md[k + y*nx] * Md[k + x*nx];
	}
	Rd[x + y*ny] = sum;
}

void correlate(int ny, int nx, const float* data, float* result) {
		
	double* dataVec;
	hipHostMalloc( (void **) &dataVec, sizeof(double)*nx*ny);
	int y, rowStart, rowEnd;
	int i;
	double meanOfEachRow, meanOfSquareRoot,num;
	std::vector<double> v(nx), vzeroSquaremean(nx);
	for(y = 0; y< ny ; ++y){        //traversing through y axis
		rowStart = y*nx;
		rowEnd = rowStart + nx;
		meanOfEachRow = std::accumulate(data+rowStart, data+rowEnd, 0.0)/nx;
		std::transform(data+rowStart, data+rowEnd, v.begin(), [&meanOfEachRow](double val){ return (val - meanOfEachRow); });
		std::transform(v.begin(), v.end(), vzeroSquaremean.begin(), [](double val){ return std::pow(val, 2); });
		meanOfSquareRoot = std::sqrt(std::accumulate(vzeroSquaremean.begin(), vzeroSquaremean.end(), 0.0));
		for(i = 0; i < nx; ++i) {
			num = v[i]/meanOfSquareRoot;
			dataVec[i+y*nx] = num;
		}
	}
	//Matrix multiplication
	//matrix nx*ny
	//matrixT ny*nx
	//after multiplication product matrix would be of size ny*ny
	dim3 dimBlock(8,8);
	dim3 dimGrid(std::ceil((double)ny/dimBlock.x), std::ceil((double)ny/(double)dimBlock.y));
	int sizeM = nx*ny*sizeof(double);
	int sizeR = ny*ny*sizeof(float);
	double* Md = 0; 
	float* Rd = 0;
	//Allocate Md and Pd on device
	hipMalloc((void **)&Md, sizeM);
	hipMemcpy(Md, dataVec, sizeM, hipMemcpyHostToDevice);
	hipMalloc((void **)&Rd, sizeR);
	matrixMul<<<dimGrid, dimBlock>>>(Md, Rd,ny, nx);
	hipMemcpy(result, Rd, sizeR, hipMemcpyDeviceToHost);

	hipFree(Md);
	hipFree(Rd);
	hipFree(dataVec);
	
}
